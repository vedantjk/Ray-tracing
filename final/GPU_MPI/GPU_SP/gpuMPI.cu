#include "hip/hip_runtime.h"

#include "gpuMPI.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define Wmax 2.0 // Window bounds
int Nrays;       // Number of rays
int n;
typedef struct
{
    float x, y, z;
} Vector;

typedef struct
{
    int i, j;
} GridPoint;

__device__ float dotProduct(Vector v1, Vector v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ Vector subtract(Vector v1, Vector v2)
{
    Vector result;
    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
    return result;
}

__device__ Vector multiply(Vector v, float scalar)
{
    Vector result;
    result.x = v.x * scalar;
    result.y = v.y * scalar;
    result.z = v.z * scalar;
    return result;
}

__device__ float magnitude(Vector v)
{
    return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ GridPoint findGridPoint(Vector W, int n)
{
    GridPoint point;
    point.i = (int)(((W.x + Wmax) / (2 * Wmax)) * n);
    point.j = (int)(((W.z + Wmax) / (2 * Wmax)) * n);
    return point;
}

__device__ Vector sampleDirection(hiprandStateXORWOW_t *state)
{
    float phi = hiprand_uniform(state) * M_PI;
    float cosTheta = hiprand_uniform(state) * 2 - 1;
    float sinTheta = sqrt(1 - cosTheta * cosTheta);
    float cosPhi = cos(phi);
    float sinPhi = sqrt(1 - cosPhi * cosPhi);
    Vector direction;
    direction.x = sinTheta * cosPhi;
    direction.y = sinTheta * sinPhi;
    direction.z = cosTheta;

    return direction;
}

__global__ void rayTracing(float *G, Vector C, Vector L, float R, float Wy, int threadsPerBlock, int numBlocks, int n, int Nrays)
{
    int ray = blockIdx.x * blockDim.x + threadIdx.x;
    long long int total = 0;
    hiprandStateXORWOW_t state;
    hiprand_init(1234, ray, 0, &state);
    long long int count = 0;
    Vector V, W, I, temp, S, N;
    float t, b;
    GridPoint gridPoint;
    while (count <= (Nrays / (threadsPerBlock * numBlocks)))
    {
        total++;
        V = sampleDirection(&state);
        W = multiply(V, Wy / V.y);
        if (abs(W.x) < Wmax && abs(W.z) < Wmax && (dotProduct(V, C) * dotProduct(V, C)) + R * R - dotProduct(C, C) > 0)
        {
            count++;
            t = dotProduct(V, C) - sqrt((dotProduct(V, C) * dotProduct(V, C)) + R * R - dotProduct(C, C));
            I = multiply(V, t);
            temp = subtract(I, C);
            N = multiply(temp, 1 / magnitude(temp));
            temp = subtract(L, I);
            S = multiply(temp, 1 / magnitude(temp));
            b = dotProduct(S, N) > 0 ? dotProduct(S, N) : 0;
            gridPoint = findGridPoint(W, n);
            // Convert 2D index to 1D index
            int index = gridPoint.i * (n + 1) + gridPoint.j;
            atomicAdd(&G[index], b);
        }
    }
    if (ray == 0)
    {
        printf("%ld\n", count);
        printf("%ld\n", total);
    }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *G, int Nrays, int n, int threadsPerBlock, int numBlocks)
{
    float *d_G;

    hipMalloc((void **)&d_G, sizeof(float) * (n + 1) * (n + 1));

    Vector C = {0, 12, 0};
    Vector L = {4, 4, -1};
    float R = 6.0;
    float Wy = 2.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    rayTracing<<<numBlocks, threadsPerBlock>>>(d_G, C, L, R, Wy, threadsPerBlock, numBlocks, n, Nrays);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("Elapsed Time: %f ms\n", milliseconds);

    hipMemcpy(G, d_G, sizeof(float) * (n + 1) * (n + 1), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    hipFree(d_G);
}
