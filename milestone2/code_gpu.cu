#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
// Elapsed Time: 544.907288 ms with 1000 80
// total: 0.939 s
#define n 1000         // Grid dimensions
#define Wmax 2.0       // Window bounds
#define Nrays (int)1e9 // Number of rays

typedef struct
{
    double x, y, z;
} Vector;

typedef struct
{
    int i, j;
} GridPoint;

__device__ double dotProduct(Vector v1, Vector v2)
{
    return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ Vector subtract(Vector v1, Vector v2)
{
    Vector result;
    result.x = v1.x - v2.x;
    result.y = v1.y - v2.y;
    result.z = v1.z - v2.z;
    return result;
}

__device__ Vector multiply(Vector v, double scalar)
{
    Vector result;
    result.x = v.x * scalar;
    result.y = v.y * scalar;
    result.z = v.z * scalar;
    return result;
}

__device__ double magnitude(Vector v)
{
    return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ GridPoint findGridPoint(Vector W)
{
    GridPoint point;
    point.i = (int)(((W.x + Wmax) / (2 * Wmax)) * n);
    point.j = (int)(((W.z + Wmax) / (2 * Wmax)) * n);
    return point;
}

__device__ Vector sampleDirection(hiprandState_t *state)
{
    double phi = hiprand_uniform(state) * M_PI;
    double cosTheta = hiprand_uniform(state)*2 - 1;
    double sinTheta = sqrt(1 - cosTheta * cosTheta);
    double cosPhi = cos(phi);
    double sinPhi = sqrt(1 - cosPhi*cosPhi);
    Vector direction;
    direction.x = sinTheta * cosPhi;
    direction.y = sinTheta * sinPhi;
    direction.z = cosTheta;

    return direction;
}

__global__ void rayTracing(double (*G)[n + 1], Vector C, Vector L, double R, double Wy, int threadsPerBlock, int numBlocks)
{
    int ray = blockIdx.x * blockDim.x + threadIdx.x;
    long long int total = 0;
    hiprandState_t state;
    hiprand_init(1234, ray, 0, &state);
    long long int count = 0;
    Vector V, W, I, temp, S, N;
    double t, b;
    GridPoint gridPoint;
    while (count < (Nrays / (threadsPerBlock * numBlocks)))
    {   
       total++;
        V = sampleDirection(&state);
        W = multiply(V, Wy / V.y);
        if (abs(W.x) < Wmax && abs(W.z) < Wmax && (dotProduct(V, C) * dotProduct(V, C)) + R * R - dotProduct(C, C) > 0)
        {
            count++;
            t = dotProduct(V, C) - sqrt((dotProduct(V, C) * dotProduct(V, C)) + R * R - dotProduct(C, C));
            I = multiply(V, t);
            temp = subtract(I, C);
            N = multiply(temp, 1 / magnitude(temp));
            temp = subtract(L, I);
            S = multiply(temp, 1 / magnitude(temp));
            b = dotProduct(S, N) > 0 ? dotProduct(S, N) : 0;
            gridPoint = findGridPoint(W);
            atomicAdd(&G[gridPoint.i][gridPoint.j], b);
        }
    }
    if(ray == 0){ 
        printf("%ld\n",count);
        printf("%ld\n",total);
    }
}

int main(int argc, char *argv[])
{
    if (argc != 3)
    {
        printf("Usage: %s <threads_per_block> <num_blocks>\n", argv[0]);
        return 1;
    }

    int threadsPerBlock = atoi(argv[1]);
    int numBlocks = atoi(argv[2]);

    double(*G)[n + 1];
    double(*d_G)[n + 1];
    hipMalloc(&d_G, sizeof(double) * (n + 1) * (n + 1));
    G = (double(*)[n + 1]) malloc(sizeof(double) * (n + 1) * (n + 1));

    Vector C = {0, 12, 0};
    Vector L = {4, 4, -1};
    double R = 6.0;
    double Wy = 2.0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    rayTracing<<<numBlocks, threadsPerBlock>>>(d_G, C, L, R, Wy, threadsPerBlock, numBlocks);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy( start );
    hipEventDestroy( stop );
    printf("Elapsed Time: %f ms\n", milliseconds);

    hipMemcpy(G, d_G, sizeof(double) * (n + 1) * (n + 1), hipMemcpyDeviceToHost);

    FILE *outputFile = fopen("output.txt", "w");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        {
            fprintf(outputFile, "%f ", G[i][j]);
        }
        fprintf(outputFile, "\n");
    }
    fclose(outputFile);

    free(G);
    hipFree(d_G);
    return 0;
}
